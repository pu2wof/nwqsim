#include "hip/hip_runtime.h"
// ---------------------------------------------------------------------------
// NWQsim: Northwest Quantum Circuit Simulation Environment
// ---------------------------------------------------------------------------
// Ang Li, Senior Computer Scientist
// Pacific Northwest National Laboratory(PNNL), U.S.
// Homepage: http://www.angliphd.com
// GitHub repo: http://www.github.com/pnnl/DM-Sim
// PNNL-IPID: 31919-E, ECCN: EAR99, IR: PNNL-SA-143160
// BSD Lincese.
// ---------------------------------------------------------------------------
// File: unit_test_nvgpu_sin.cuh
// Unit test for single GPU state-vector simulation with NVIDIA GPU backend.
// ---------------------------------------------------------------------------

#include <stdio.h>
#include <mpi.h>
#include "util.h"
#include "dmsim_nvgpu_mpi.cuh"

using namespace NWQSim;
using namespace std;
#define TEST(X) pass = pass && X;

bool check_sv(Simulation& sim, ValType* sv_real_expected, 
        ValType* sv_imag_expected)
{
    bool pass = true;
    for (int i=0; i<sim.dim; i++)
    {
        ValType real_diff = abs(sv_real_expected[i] - sim.sv_real_cpu[i]);
        ValType imag_diff = abs(sv_imag_expected[i] - sim.sv_imag_cpu[i]);
        //printf("(%lf,%lf)",real_diff, imag_diff);
        if ( real_diff > ERROR_BAR || imag_diff > ERROR_BAR)
        {
            pass = false;
            break;
        }
    }
    return pass;
}


//============== X Gate ================
bool test_X()
{
    const int n_qubits = 1;
    const int dim = ((IdxType)1<<(2*n_qubits));
    bool pass = true;
    Simulation sim;
    sim.AllocateQubit();
    sim.X(0);
    sim.sim();
    //sim.print_res_sv();
    ValType sv_real_expected[dim] = {0,0,0,1};
    ValType sv_imag_expected[dim] = {0,0,0,0};
    pass = check_sv(sim, sv_real_expected, sv_imag_expected);
    std::cout << "X gate test: " << (pass?"Success":"Failed") << std::endl;
    return pass;
}

bool test_CX()
{
    bool pass = true;
    Simulation sim;
    //sim.AllocateQubit();
    //sim.AllocateQubit();
    //sim.AllocateQubit();
    //sim.AllocateQubit();
    //sim.AllocateQubit();
    //sim.AllocateQubit();
    //sim.AllocateQubit();

    //sim.ID(0);




    //sim.X(5);
    //sim.SX(6);
    //sim.RZ(PI/7,6);
    //sim.SX(6);
    //sim.CX(5,6);



    
    //sim.X(2);
    //sim.X(3);
    //sim.X(4);
    //sim.X(5);
    //sim.X(6);

    //sim.ID(1);
    //sim.X(0);
    //sim.SX(0);
    //sim.RZ(PI/7,0);

    //==== For test 1-q fusion
    //sim.X(0);
    //sim.SX(0);
    //sim.X(2);
    //sim.X(3);
    //sim.RZ(PI/3,0);
    //sim.CX(0,5);
    //sim.X(2);
    //sim.RZ(PI/9,0);
    //sim.SX(5);
    //sim.SX(0);

    //==== For test 2-q fusion
    //sim.X(0);
    //sim.SX(3);
    //sim.CX(0,3);
    //sim.X(4);
    //sim.SX(0);
    //sim.RZ(PI/7,3);
    //sim.CX(0,3);
    //sim.CX(3,0);
    //sim.CX(0,3);


    sim.X(0);
    sim.CX(0,2);
    sim.CX(2,1);
    sim.CX(0,2);

    //sim.X(4);

    //sim.X(4);
    //sim.X(5);
    //sim.X(6);

    //sim.CX(1,0);
    //sim.CX(2,1);
    //sim.CX(3,2);

    //sim.CX(0,4);


    //sim.CX(5,4);


    IdxType* res = sim.measure_all(10);
    print_measurement(res,8,10);


    //sim.CX(5,4);
    //sim.CX(6,5);

    //sim.M(0);

    //sim.sim();
    //sim.print_res_sv();

    //IdxType res0 = sim.measure(0);
    //std::cout << "=== Measure result q-0: " << res0 << std::endl;
    //sim.print_res_sv();
    //IdxType res1 = sim.measure(1);
    //std::cout << "=== Measure result q-1: " << res1 << std::endl;
    //sim.print_res_sv();



    //ValType sv_real_expected[dim] = {0.5,0,0,0.5, 0,0,0,0, 0,0,0,0, 0.5,0,0,0.5};
    //ValType sv_imag_expected[dim] = {0};
    //pass = check_sv(sim, sv_real_expected, sv_imag_expected);
    //std::cout << "CX gate test: " << (pass?"Success":"Failed") << std::endl;
    return pass;
}


int main(int argc, char *argv[])
{
    MPI_Init(&argc, &argv);
//=================================== Initialization =====================================
    bool pass = true;

    //TEST(test_X());
    TEST(test_CX());
    //std::cout << "\nUnit Test for NVGPU SIN " 
    //<< (pass?"Success":"Failed") << " !!!" << std::endl;
    



    MPI_Finalize();

    return 0;
}

