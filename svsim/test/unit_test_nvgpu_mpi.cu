#include "hip/hip_runtime.h"
// ---------------------------------------------------------------------------
// NWQsim: Northwest Quantum Circuit Simulation Environment
// ---------------------------------------------------------------------------
// Ang Li, Senior Computer Scientist
// Pacific Northwest National Laboratory(PNNL), U.S.
// Homepage: http://www.angliphd.com
// GitHub repo: http://www.github.com/pnnl/DM-Sim
// PNNL-IPID: 31919-E, ECCN: EAR99, IR: PNNL-SA-143160
// BSD Lincese.
// ---------------------------------------------------------------------------
// File: unit_test_nvgpu_sin.cuh
// Unit test for single GPU state-vector simulation with NVIDIA GPU backend.
// ---------------------------------------------------------------------------

#include <stdio.h>
#include <mpi.h>
#include "util.h"
#include "svsim_nvgpu_mpi.cuh"

using namespace NWQSim;
using namespace std;
#define TEST(X) pass = pass && X;

bool check_sv(Simulation& sim, ValType* sv_real_expected, 
        ValType* sv_imag_expected)
{
    bool pass = true;
    for (int i=0; i<sim.dim; i++)
    {
        ValType real_diff = abs(sv_real_expected[i] - sim.sv_real_cpu[i]);
        ValType imag_diff = abs(sv_imag_expected[i] - sim.sv_imag_cpu[i]);
        //printf("(%lf,%lf)",real_diff, imag_diff);
        if ( real_diff > ERROR_BAR || imag_diff > ERROR_BAR)
        {
            pass = false;
            break;
        }
    }
    return pass;
}


//============== X Gate ================
bool test_X()
{
    const int n_qubits = 4;
    const int dim = ((IdxType)1<<(n_qubits));
    bool pass = true;
    Simulation sim(n_qubits);

    sim.X(0);
    sim.CX(0,1);
    sim.SX(1);
    sim.RX(PI/8,0);
    sim.RY(PI/9,1);
    sim.H(1);
    sim.P(PI/5,0);
    sim.CX(1,0);
    sim.H(0);



    //sim.M(0);
    //sim.RESET(0);
    //sim.RESET(1);
    // 
    
    //sim.CU(PI/6,PI/7,PI/8,PI/9,0,1);
    //sim.Y(0);
    //sim.SX(0);
    //sim.S(0);
    //sim.T(0);

    sim.sim();
    sim.print_res_sv();
    //ValType sv_real_expected[dim] = {0,1};
    //ValType sv_imag_expected[dim] = {0,0};
    //pass = check_sv(sim, sv_real_expected, sv_imag_expected);
    //std::cout << "X gate test: " << (pass?"Success":"Failed") << std::endl;
    return pass;
}

int main(int argc, char *argv[])
{
    MPI_Init(&argc, &argv);
//=================================== Initialization =====================================
    bool pass = true;

    TEST(test_X());

    //std::cout << "\nUnit Test for NVGPU SIN " 
    //<< (pass?"Success":"Failed") << " !!!" << std::endl;


    MPI_Finalize();

    return 0;
}

